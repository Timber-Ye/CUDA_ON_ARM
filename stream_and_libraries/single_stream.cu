#include "hip/hip_runtime.h"
//
// Created by hanchiao on 2/9/23.
//
#include <cstdio>
#include "../matmul/error.cuh"

#define N (1024*1024)
#define FULL_DATA_SIZE   (N*20)

__global__ void kernel( int *a, int *b, int *c ) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        int idx1 = (idx + 1) % 256;
        int idx2 = (idx + 2) % 256;
        float   as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
        float   bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
        c[idx] = (as + bs) / 2;
    }
}

int main(void){
    hipDeviceProp_t  prop;
    int whichDevice;
    CHECK( hipGetDevice( &whichDevice ) );
    CHECK( hipGetDeviceProperties( &prop, whichDevice ) );
    if (!prop.deviceOverlap) {
        printf( "Device will not handle overlaps, so no speed up from streams\n" );
        return 0;
    }

    int *h_a, *h_b, *h_c;
    int *d_a_0, *d_b_0, *d_c_0;

    hipStream_t stream_0;
    CHECK(hipStreamCreate(&stream_0));

    hipEvent_t     start, stop;
    float           elapsedTime;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

    CHECK(hipHostAlloc((void **)&h_a, sizeof(int) * FULL_DATA_SIZE, hipHostMallocDefault));
    CHECK(hipHostAlloc((void **)&h_b, sizeof(int) * FULL_DATA_SIZE, hipHostMallocDefault));
    CHECK(hipHostAlloc((void **)&h_c, sizeof(int) * FULL_DATA_SIZE, hipHostMallocDefault));

    CHECK(hipMalloc((void **)&d_a_0, sizeof(int) * N));
    CHECK(hipMalloc((void **)&d_b_0, sizeof(int) * N));
    CHECK(hipMalloc((void **)&d_c_0, sizeof(int) * N));

    for (int i=0; i<FULL_DATA_SIZE; i++) {
        h_a[i] = rand();
        h_b[i] = rand();
    }

    CHECK(hipEventRecord(start, 0));
    for(int i=0; i<FULL_DATA_SIZE; i+= N){
        CHECK(hipMemcpyAsync(d_a_0, h_a+i, sizeof(int) * N, hipMemcpyHostToDevice, stream_0));

        CHECK(hipMemcpyAsync(d_b_0, h_b+i, sizeof(int) * N, hipMemcpyHostToDevice, stream_0));

        kernel<<<N/256,1024,0,stream_0>>>( d_a_0, d_b_0, d_c_0 );

        CHECK(hipMemcpyAsync(h_c+i, d_c_0, N * sizeof(int), hipMemcpyDeviceToHost, stream_0));
    }

    CHECK(hipStreamSynchronize(stream_0));
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&elapsedTime, start, stop));

    printf("Time consuming: %3.1f ms\n", elapsedTime);

    // cleanup the streams and memory
    CHECK( hipHostFree( h_a ) );
    CHECK( hipHostFree( h_b ) );
    CHECK( hipHostFree( h_c ) );
    CHECK( hipFree( d_a_0 ) );
    CHECK( hipFree( d_b_0 ) );
    CHECK( hipFree( d_c_0 ) );
    CHECK( hipStreamDestroy( stream_0 ) );

    return 0;
}
