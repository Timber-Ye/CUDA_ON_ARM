#include "hip/hip_runtime.h"
//
// Created by hanchiao on 2/11/23.
//

#include <stdio.h>
#include <stdlib.h>
#include "../matmul/error.cuh"

#define N 3000 // Love u 3000 times!
#define BLOCK_SIZE 32

__managed__ int input_Matrix[N][N];
__managed__ int output_GPU[N][N];
__managed__ int output_CPU[N][N];
__global__ void huanhuanhuanhuanhuan(int input_M[N][N], int output_M[N][N])
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    if(x < N && y < N){
        output_M[y][x] = input_M[y][x] * 5;
    }
}

void cpu_huanhuanhuanhuanhuan(int intput_M[N][N], int output_CPU[N][N])
{

    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            output_CPU[i][j] = intput_M[i][j] * 5;
        }
    }
}

int main(int argc, char const* argv[])
{

    hipEvent_t start, stop_gpu;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop_gpu));


    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j)
        {

            input_Matrix[i][j] = rand() % 3001;
            //printf("%d ",input_Matrix[i][j]);
        }
        //printf("\n");
    }
    cpu_huanhuanhuanhuanhuan(input_Matrix, output_CPU);

    CHECK(hipEventRecord(start));
    unsigned int grid_rows = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    printf("\n***********GPU RUN**************\n");
    huanhuanhuanhuanhuan <<<dimGrid, dimBlock >>> (input_Matrix, output_GPU);
    CHECK(hipDeviceSynchronize());
    CHECK(hipEventRecord(stop_gpu));
    CHECK(hipEventSynchronize(stop_gpu));

    float elapsed_time_gpu;
    CHECK(hipEventElapsedTime(&elapsed_time_gpu, start, stop_gpu));
    printf("Time_GPU = %g ms.\n", elapsed_time_gpu);

    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop_gpu));

    int ok = 1;
    for (int i = 0; i < N; ++i)
    {
        for (int j = 0; j < N; ++j)
        {
            //printf("%d ",output_GPU[i][j]);
            if (fabs(output_GPU[i][j] - output_CPU[i][j]) > (1.0e-10))
            {
                ok = 0;
            }

        }
        //printf("\n");
    }
    printf("\n***********Check result**************\n");
    for (int i = 0; i < N; ++i)
    {
        for (int j = 0; j < N; ++j)
        {
            //printf("%d ",output_CPU[i][j]);
            if (fabs(output_GPU[i][j] - output_CPU[i][j]) > (1.0e-10))
            {
                ok = 0;
            }

        }
        //printf("\n");
    }


    if (ok)
    {
        printf("Pass!!!\n");
    }
    else
    {
        printf("Error!!!\n");
    }

    // free memory
    return 0;
}
